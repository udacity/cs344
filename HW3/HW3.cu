#include "hip/hip_runtime.h"
#include "utils.h"
#include <string>
#include "loadSaveImage.h"
#include <thrust/extrema.h>

//chroma-LogLuminance Space
static float *d_x__;
static float *d_y__;
static float *d_logY__;

//memory for the cdf
static unsigned int *d_cdf__;

static const int numBins = 1024;

size_t numRows__;
size_t numCols__;

/* Copied from Mike's IPython notebook with some minor modifications
 * Mainly double precision constants to floats and log10 -> log10f
 * Also removed Luminance (Y) channel since it is never used       eke*/

__global__ void rgb_to_xyY(
    float* d_r,
    float* d_g,
    float* d_b,
    float* d_x,
    float* d_y,
    float* d_log_Y,
    float  delta,
    int    num_pixels_y,
    int    num_pixels_x )
{
  int  ny             = num_pixels_y;
  int  nx             = num_pixels_x;
  int2 image_index_2d = make_int2( ( blockIdx.x * blockDim.x ) + threadIdx.x, ( blockIdx.y * blockDim.y ) + threadIdx.y );
  int  image_index_1d = ( nx * image_index_2d.y ) + image_index_2d.x;

  if ( image_index_2d.x < nx && image_index_2d.y < ny )
  {
    float r = d_r[ image_index_1d ];
    float g = d_g[ image_index_1d ];
    float b = d_b[ image_index_1d ];

    float X = ( r * 0.4124f ) + ( g * 0.3576f ) + ( b * 0.1805f );
    float Y = ( r * 0.2126f ) + ( g * 0.7152f ) + ( b * 0.0722f );
    float Z = ( r * 0.0193f ) + ( g * 0.1192f ) + ( b * 0.9505f );

    float L = X + Y + Z;
    float x = X / L;
    float y = Y / L;

    float log_Y = log10f( delta + Y );

    d_x[ image_index_1d ]     = x;
    d_y[ image_index_1d ]     = y;
    d_log_Y[ image_index_1d ] = log_Y;
  }
}

/* Copied from Mike's IPython notebook *
   Modified just by having threads read the 
   normalization constant directly from device memory
   instead of copying it back                          */


__global__ void normalize_cdf(
    unsigned int* d_input_cdf,
    float*        d_output_cdf,
    int           n
    )
{
  const float normalization_constant = 1.f / d_input_cdf[n - 1];

  int global_index_1d = ( blockIdx.x * blockDim.x ) + threadIdx.x;

  if ( global_index_1d < n )
  {
    unsigned int input_value  = d_input_cdf[ global_index_1d ];
    float        output_value = input_value * normalization_constant;

    d_output_cdf[ global_index_1d ] = output_value;
  }
}


/* Copied from Mike's IPython notebook *
   Modified double constants -> float  *
   Perform tone mapping based upon new *
   luminance scaling                   */

__global__ void tonemap(
    float* d_x,
    float* d_y,
    float* d_log_Y,
    float* d_cdf_norm,
    float* d_r_new,
    float* d_g_new,
    float* d_b_new,
    float  min_log_Y,
    float  max_log_Y,
    float  log_Y_range,
    int    num_bins,
    int    num_pixels_y,
    int    num_pixels_x )
{
  int  ny             = num_pixels_y;
  int  nx             = num_pixels_x;
  int2 image_index_2d = make_int2( ( blockIdx.x * blockDim.x ) + threadIdx.x, ( blockIdx.y * blockDim.y ) + threadIdx.y );
  int  image_index_1d = ( nx * image_index_2d.y ) + image_index_2d.x;

  if ( image_index_2d.x < nx && image_index_2d.y < ny )
  {
    float x         = d_x[ image_index_1d ];
    float y         = d_y[ image_index_1d ];
    float log_Y     = d_log_Y[ image_index_1d ];
    int   bin_index = min( num_bins - 1, int( (num_bins * ( log_Y - min_log_Y ) ) / log_Y_range ) );
    float Y_new     = d_cdf_norm[ bin_index ];

    float X_new = x * ( Y_new / y );
    float Z_new = ( 1 - x - y ) * ( Y_new / y );

    float r_new = ( X_new *  3.2406f ) + ( Y_new * -1.5372f ) + ( Z_new * -0.4986f );
    float g_new = ( X_new * -0.9689f ) + ( Y_new *  1.8758f ) + ( Z_new *  0.0415f );
    float b_new = ( X_new *  0.0557f ) + ( Y_new * -0.2040f ) + ( Z_new *  1.0570f );

    d_r_new[ image_index_1d ] = r_new;
    d_g_new[ image_index_1d ] = g_new;
    d_b_new[ image_index_1d ] = b_new;
  }
}


//return types are void since any internal error will be handled by quitting
//no point in returning error codes...
void preProcess(float** d_luminance, unsigned int** d_cdf,
                size_t *numRows, size_t *numCols,
                unsigned int *numberOfBins,
                const std::string &filename) {
  //make sure the context initializes ok
  checkCudaErrors(hipFree(0));

  float *imgPtr; //we will become responsible for this pointer
  loadImageHDR(filename, &imgPtr, &numRows__, &numCols__);
  *numRows = numRows__;
  *numCols = numCols__;

  //first thing to do is split incoming BGR float data into separate channels
  size_t numPixels = numRows__ * numCols__;
  float *red   = new float[numPixels];
  float *green = new float[numPixels];
  float *blue  = new float[numPixels];

  //Remeber image is loaded BGR
  for (size_t i = 0; i < numPixels; ++i) {
    blue[i]  = imgPtr[3 * i + 0];
    green[i] = imgPtr[3 * i + 1];
    red[i]   = imgPtr[3 * i + 2];
  }

  delete[] imgPtr; //being good citizens are releasing resources
                   //allocated in loadImageHDR

  float *d_red, *d_green, *d_blue;  //RGB space

  size_t channelSize = sizeof(float) * numPixels;

  checkCudaErrors(hipMalloc(&d_red,    channelSize));
  checkCudaErrors(hipMalloc(&d_green,  channelSize));
  checkCudaErrors(hipMalloc(&d_blue,   channelSize));
  checkCudaErrors(hipMalloc(&d_x__,    channelSize));
  checkCudaErrors(hipMalloc(&d_y__,    channelSize));
  checkCudaErrors(hipMalloc(&d_logY__, channelSize));

  checkCudaErrors(hipMemcpy(d_red,   red,   channelSize, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_green, green, channelSize, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_blue,  blue,  channelSize, hipMemcpyHostToDevice));

  //convert from RGB space to chrominance/luminance space xyY
  const dim3 blockSize(32, 16, 1);
  const dim3 gridSize( (numCols__ + blockSize.x - 1) / blockSize.x, 
                       (numRows__ + blockSize.y - 1) / blockSize.y, 1);
  rgb_to_xyY<<<gridSize, blockSize>>>(d_red, d_green, d_blue,
                                      d_x__, d_y__,   d_logY__,
                                      .0001f, numRows__, numCols__);

  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

  *d_luminance = d_logY__;

  //allocate memory for the cdf of the histogram
  *numberOfBins = numBins;
  checkCudaErrors(hipMalloc(&d_cdf__, sizeof(unsigned int) * numBins));
  checkCudaErrors(hipMemset(d_cdf__, 0, sizeof(unsigned int) * numBins));

  *d_cdf = d_cdf__;

  checkCudaErrors(hipFree(d_red));
  checkCudaErrors(hipFree(d_green));
  checkCudaErrors(hipFree(d_blue));

  delete[] red;
  delete[] green;
  delete[] blue;
}

void postProcess(const std::string& output_file, 
                 size_t numRows, size_t numCols,
                 float min_log_Y, float max_log_Y) {
  const int numPixels = numRows__ * numCols__;

  const int numThreads = 192;

  float *d_cdf_normalized;

  checkCudaErrors(hipMalloc(&d_cdf_normalized, sizeof(float) * numBins));

  //first normalize the cdf to a maximum value of 1
  //this is how we compress the range of the luminance channel
  normalize_cdf<<< (numBins + numThreads - 1) / numThreads,
                    numThreads>>>(d_cdf__,
                                  d_cdf_normalized,
                                  numBins);

  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

  //allocate memory for the output RGB channels
  float *h_red, *h_green, *h_blue;
  float *d_red, *d_green, *d_blue;

  h_red   = new float[numPixels];
  h_green = new float[numPixels];
  h_blue  = new float[numPixels];

  checkCudaErrors(hipMalloc(&d_red,   sizeof(float) * numPixels));
  checkCudaErrors(hipMalloc(&d_green, sizeof(float) * numPixels));
  checkCudaErrors(hipMalloc(&d_blue,  sizeof(float) * numPixels));

  float log_Y_range = max_log_Y - min_log_Y;

  const dim3 blockSize(32, 16, 1);
  const dim3 gridSize( (numCols + blockSize.x - 1) / blockSize.x,
                       (numRows + blockSize.y - 1) / blockSize.y );
  //next perform the actual tone-mapping
  //we map each luminance value to its new value
  //and then transform back to RGB space
  tonemap<<<gridSize, blockSize>>>(d_x__, d_y__, d_logY__,
                                   d_cdf_normalized,
                                   d_red, d_green, d_blue,
                                   min_log_Y, max_log_Y,
                                   log_Y_range, numBins,
                                   numRows, numCols);

  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

  checkCudaErrors(hipMemcpy(h_red,   d_red,   sizeof(float) * numPixels, hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_green, d_green, sizeof(float) * numPixels, hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_blue,  d_blue,  sizeof(float) * numPixels, hipMemcpyDeviceToHost));

  //recombine the image channels
  float *imageHDR = new float[numPixels * 3];

  for (int i = 0; i < numPixels; ++i) {
    imageHDR[3 * i + 0] = h_blue[i];
    imageHDR[3 * i + 1] = h_green[i];
    imageHDR[3 * i + 2] = h_red[i];
  }

  saveImageHDR(imageHDR, numRows, numCols, output_file);

  delete[] imageHDR;
  delete[] h_red;
  delete[] h_green;
  delete[] h_blue;

  //cleanup
  checkCudaErrors(hipFree(d_cdf_normalized));
}

void cleanupGlobalMemory(void)
{
  checkCudaErrors(hipFree(d_x__));
  checkCudaErrors(hipFree(d_y__));
  checkCudaErrors(hipFree(d_logY__));
  checkCudaErrors(hipFree(d_cdf__));
}
