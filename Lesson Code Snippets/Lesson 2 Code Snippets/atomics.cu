#include "hip/hip_runtime.h"
#include <stdio.h>
#include "gputimer.h"

#define NUM_THREADS 1000000
#define ARRAY_SIZE  100

#define BLOCK_WIDTH 1000

void print_array(int *array, int size)
{
    printf("{ ");
    for (int i = 0; i < size; i++)  { printf("%d ", array[i]); }
    printf("}\n");
}

__global__ void increment_naive(int *g)
{
	// which thread is this?
	int i = blockIdx.x * blockDim.x + threadIdx.x; 

	// each thread to increment consecutive elements, wrapping at ARRAY_SIZE
	i = i % ARRAY_SIZE;  
	g[i] = g[i] + 1;
}

__global__ void increment_atomic(int *g)
{
	// which thread is this?
	int i = blockIdx.x * blockDim.x + threadIdx.x; 

	// each thread to increment consecutive elements, wrapping at ARRAY_SIZE
	i = i % ARRAY_SIZE;  
	atomicAdd(& g[i], 1);
}

int main(int argc,char **argv)
{   
    GpuTimer timer;
    printf("%d total threads in %d blocks writing into %d array elements\n",
           NUM_THREADS, NUM_THREADS / BLOCK_WIDTH, ARRAY_SIZE);

    // declare and allocate host memory
    int *h_array = new int[ARRAY_SIZE];
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(int);
 
    // declare, allocate, and zero out GPU memory
    int * d_array;
    hipMalloc((void **) &d_array, ARRAY_BYTES);
    hipMemset((void *) d_array, 0, ARRAY_BYTES); 

    // launch the kernel - comment out one of these
    timer.Start();
    // increment_naive<<<NUM_THREADS/BLOCK_WIDTH, BLOCK_WIDTH>>>(d_array);
    increment_atomic<<<NUM_THREADS/BLOCK_WIDTH, BLOCK_WIDTH>>>(d_array);
    timer.Stop();
    
    // copy back the array of sums from GPU and print
    hipMemcpy(h_array, d_array, ARRAY_BYTES, hipMemcpyDeviceToHost);
    print_array(h_array, ARRAY_SIZE);
    printf("Time elapsed = %g ms\n", timer.Elapsed());
 
    // free GPU memory allocation and exit
    hipFree(d_array);
    return 0;
}
