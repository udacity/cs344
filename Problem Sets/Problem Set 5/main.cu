#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <cstdio>
#include <fstream>
#include "utils.h"
#include "timer.h"
#include <cstdio>
#if defined(_WIN16) || defined(_WIN32) || defined(_WIN64)
#include <Windows.h>
#else
#include <sys/time.h>
#endif

#include <thrust/random/linear_congruential_engine.h>
#include <thrust/random/normal_distribution.h>
#include <thrust/random/uniform_int_distribution.h>

#include "reference_calc.h"

void computeHistogram(const unsigned int *const d_vals,
                      unsigned int* const d_histo,
                      const unsigned int numBins,
                      const unsigned int numElems);

int main(void)
{
  const unsigned int numBins = 1024;
  const unsigned int numElems = 10000 * numBins;
  const float stddev = 100.f;

  unsigned int *vals = new unsigned int[numElems];
  unsigned int *h_vals = new unsigned int[numElems];
  unsigned int *h_studentHisto = new unsigned int[numBins];
  unsigned int *h_refHisto = new unsigned int[numBins];

#if defined(_WIN16) || defined(_WIN32) || defined(_WIN64)
  srand(GetTickCount());
#else
  timeval tv;
  gettimeofday(&tv, NULL);

  srand(tv.tv_usec);
#endif

  //make the mean unpredictable, but close enough to the middle
  //so that timings are unaffected
  unsigned int mean = rand() % 100 + 462;

  //Output mean so that grading can happen with the same inputs
  std::cout << mean << std::endl;

  thrust::minstd_rand rng;

  thrust::random::normal_distribution<float> normalDist((float)mean, stddev);

  // Generate the random values
  for (size_t i = 0; i < numElems; ++i) {
    vals[i] = std::min((unsigned int) std::max((int)normalDist(rng), 0), numBins - 1);
  }

  unsigned int *d_vals, *d_histo;

  GpuTimer timer;

  checkCudaErrors(hipMalloc(&d_vals,    sizeof(unsigned int) * numElems));
  checkCudaErrors(hipMalloc(&d_histo,   sizeof(unsigned int) * numBins));
  checkCudaErrors(hipMemset(d_histo, 0, sizeof(unsigned int) * numBins));

  checkCudaErrors(hipMemcpy(d_vals, vals, sizeof(unsigned int) * numElems, hipMemcpyHostToDevice));

  timer.Start();
  computeHistogram(d_vals, d_histo, numBins, numElems);
  timer.Stop();
  int err = printf("Your code ran in: %f msecs.\n", timer.Elapsed());

  if (err < 0) {
    //Couldn't print! Probably the student closed stdout - bad news
    std::cerr << "Couldn't print timing information! STDOUT Closed!" << std::endl;
    exit(1);
  }

  // copy the student-computed histogram back to the host
  checkCudaErrors(hipMemcpy(h_studentHisto, d_histo, sizeof(unsigned int) * numBins, hipMemcpyDeviceToHost));

  //generate reference for the given mean
  reference_calculation(vals, h_refHisto, numBins, numElems);

  //Now do the comparison
  checkResultsExact(h_refHisto, h_studentHisto, numBins);

  delete[] h_vals;
  delete[] h_refHisto;
  delete[] h_studentHisto;

  hipFree(d_vals);
  hipFree(d_histo);

  return 0;
}
