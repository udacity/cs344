#include "hip/hip_runtime.h"
/* Udacity Homework 3
   HDR Tone-mapping

  Background HDR
  ==============

  A High Dynamic Range (HDR) image contains a wider variation of intensity
  and color than is allowed by the RGB format with 1 byte per channel that we
  have used in the previous assignment.  

  To store this extra information we use single precision floating point for
  each channel.  This allows for an extremely wide range of intensity values.

  In the image for this assignment, the inside of church with light coming in
  through stained glass windows, the raw input floating point values for the
  channels range from 0 to 275.  But the mean is .41 and 98% of the values are
  less than 3!  This means that certain areas (the windows) are extremely bright
  compared to everywhere else.  If we linearly map this [0-275] range into the
  [0-255] range that we have been using then most values will be mapped to zero!
  The only thing we will be able to see are the very brightest areas - the
  windows - everything else will appear pitch black.

  The problem is that although we have cameras capable of recording the wide
  range of intensity that exists in the real world our monitors are not capable
  of displaying them.  Our eyes are also quite capable of observing a much wider
  range of intensities than our image formats / monitors are capable of
  displaying.

  Tone-mapping is a process that transforms the intensities in the image so that
  the brightest values aren't nearly so far away from the mean.  That way when
  we transform the values into [0-255] we can actually see the entire image.
  There are many ways to perform this process and it is as much an art as a
  science - there is no single "right" answer.  In this homework we will
  implement one possible technique.

  Background Chrominance-Luminance
  ================================

  The RGB space that we have been using to represent images can be thought of as
  one possible set of axes spanning a three dimensional space of color.  We
  sometimes choose other axes to represent this space because they make certain
  operations more convenient.

  Another possible way of representing a color image is to separate the color
  information (chromaticity) from the brightness information.  There are
  multiple different methods for doing this - a common one during the analog
  television days was known as Chrominance-Luminance or YUV.

  We choose to represent the image in this way so that we can remap only the
  intensity channel and then recombine the new intensity values with the color
  information to form the final image.

  Old TV signals used to be transmitted in this way so that black & white
  televisions could display the luminance channel while color televisions would
  display all three of the channels.
  

  Tone-mapping
  ============

  In this assignment we are going to transform the luminance channel (actually
  the log of the luminance, but this is unimportant for the parts of the
  algorithm that you will be implementing) by compressing its range to [0, 1].
  To do this we need the cumulative distribution of the luminance values.

  Example
  -------

  input : [2 4 3 3 1 7 4 5 7 0 9 4 3 2]
  min / max / range: 0 / 9 / 9

  histo with 3 bins: [4 7 3]

  cdf : [4 11 14]


  Your task is to calculate this cumulative distribution by following these
  steps.

*/

#include <limits.h>
#include <float.h>
#include <math.h>
#include <stdio.h>

#include "utils.h"

__global__
void histogram_kernel(unsigned int* d_bins, const float* d_in, const int bin_count, const float lum_min, const float lum_max, const int size) {  
    int mid = threadIdx.x + blockDim.x * blockIdx.x;
    if(mid >= size)
        return;
    float lum_range = lum_max - lum_min;
    int bin = ((d_in[mid]-lum_min) / lum_range) * bin_count;
    
    atomicAdd(&d_bins[bin], 1);
}

__global__ 
void scan_kernel(unsigned int* d_bins, int size) {
    int mid = threadIdx.x + blockDim.x * blockIdx.x;
    if(mid >= size)
        return;
    
    for(int s = 1; s <= size; s *= 2) {
          int spot = mid - s; 
         
          unsigned int val = 0;
          if(spot >= 0)
              val = d_bins[spot];
          __syncthreads();
          if(spot >= 0)
              d_bins[mid] += val;
          __syncthreads();

    }
}
// calculate reduce max or min and stick the value in d_answer.
__global__
void reduce_minmax_kernel(const float* const d_in, float* d_out, const size_t size, int minmax) {
    extern __shared__ float shared[];
    
    int mid = threadIdx.x + blockDim.x * blockIdx.x;
    int tid = threadIdx.x; 
    
    // we have 1 thread per block, so copying the entire block should work fine
    if(mid < size) {
        shared[tid] = d_in[mid];
    } else {
        if(minmax == 0)
            shared[tid] = FLT_MAX;
        else
            shared[tid] = -FLT_MAX;
    }
    
    // wait for all threads to copy the memory
    __syncthreads();
    
    // don't do any thing with memory if we happen to be far off ( I don't know how this works with
    // sync threads so I moved it after that point )
    if(mid >= size) {   
        if(tid == 0) {
            if(minmax == 0) 
                d_out[blockIdx.x] = FLT_MAX;
            else
                d_out[blockIdx.x] = -FLT_MAX;

        }
        return;
    }
       
    for(unsigned int s = blockDim.x/2; s > 0; s /= 2) {
        if(tid < s) {
            if(minmax == 0) {
                shared[tid] = min(shared[tid], shared[tid+s]);
            } else {
                shared[tid] = max(shared[tid], shared[tid+s]);
            }
        }
        
        __syncthreads();
    }
    
    if(tid == 0) {
        d_out[blockIdx.x] = shared[0];
    }
}

int get_max_size(int n, int d) {
    return (int)ceil( (float)n/(float)d ) + 1;
}

float reduce_minmax(const float* const d_in, const size_t size, int minmax) {
    int BLOCK_SIZE = 32;
    // we need to keep reducing until we get to the amount that we consider 
    // having the entire thing fit into one block size
    size_t curr_size = size;
    float* d_curr_in;
    
    checkCudaErrors(hipMalloc(&d_curr_in, sizeof(float) * size));    
    checkCudaErrors(hipMemcpy(d_curr_in, d_in, sizeof(float) * size, hipMemcpyDeviceToDevice));


    float* d_curr_out;
    
    dim3 thread_dim(BLOCK_SIZE);
    const int shared_mem_size = sizeof(float)*BLOCK_SIZE;
    
    while(1) {
        checkCudaErrors(hipMalloc(&d_curr_out, sizeof(float) * get_max_size(curr_size, BLOCK_SIZE)));
        
        dim3 block_dim(get_max_size(size, BLOCK_SIZE));
        reduce_minmax_kernel<<<block_dim, thread_dim, shared_mem_size>>>(
            d_curr_in,
            d_curr_out,
            curr_size,
            minmax
        );
        hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

            
        // move the current input to the output, and clear the last input if necessary
        checkCudaErrors(hipFree(d_curr_in));
        d_curr_in = d_curr_out;
        
        if(curr_size <  BLOCK_SIZE) 
            break;
        
        curr_size = get_max_size(curr_size, BLOCK_SIZE);
    }
    
    // theoretically we should be 
    float h_out;
    hipMemcpy(&h_out, d_curr_out, sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_curr_out);
    return h_out;
}

void your_histogram_and_prefixsum(const float* const d_logLuminance,
                                  unsigned int* const d_cdf,
                                  float &min_logLum,
                                  float &max_logLum,
                                  const size_t numRows,
                                  const size_t numCols,
                                  const size_t numBins)
{
    const size_t size = numRows*numCols;
    min_logLum = reduce_minmax(d_logLuminance, size, 0);
    max_logLum = reduce_minmax(d_logLuminance, size, 1);
    
    printf("got min of %f\n", min_logLum);
    printf("got max of %f\n", max_logLum);
    printf("numBins %d\n", numBins);
    
    unsigned int* d_bins;
    size_t histo_size = sizeof(unsigned int)*numBins;

    checkCudaErrors(hipMalloc(&d_bins, histo_size));    
    checkCudaErrors(hipMemset(d_bins, 0, histo_size));  
    dim3 thread_dim(1024);
    dim3 hist_block_dim(get_max_size(size, thread_dim.x));
    histogram_kernel<<<hist_block_dim, thread_dim>>>(d_bins, d_logLuminance, numBins, min_logLum, max_logLum, size);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

    unsigned int h_out[100];
    hipMemcpy(&h_out, d_bins, sizeof(unsigned int)*100, hipMemcpyDeviceToHost);
    for(int i = 0; i < 100; i++)
        printf("hist out %d\n", h_out[i]);
    
    dim3 scan_block_dim(get_max_size(numBins, thread_dim.x));

    scan_kernel<<<scan_block_dim, thread_dim>>>(d_bins, numBins);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    
    hipMemcpy(&h_out, d_bins, sizeof(unsigned int)*100, hipMemcpyDeviceToHost);
    for(int i = 0; i < 100; i++)
        printf("cdf out %d\n", h_out[i]);
    

    hipMemcpy(d_cdf, d_bins, histo_size, hipMemcpyDeviceToDevice);

    
    checkCudaErrors(hipFree(d_bins));
     
    
  //TODO
  /*Here are the steps you need to implement
    1) find the minimum and maximum value in the input logLuminance channel
       store in min_logLum and max_logLum
    2) subtract them to find the range
    3) generate a histogram of all the values in the logLuminance channel using
       the formula: bin = (lum[i] - lumMin) / lumRange * numBins
    4) Perform an exclusive scan (prefix sum) on the histogram to get
       the cumulative distribution of luminance values (this should go in the
       incoming d_cdf pointer which already has been allocated for you)       */
}
