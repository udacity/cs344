#include "hip/hip_runtime.h"
//Udacity HW 6
//Poisson Blending

/* Background
   ==========

   The goal for this assignment is to take one image (the source) and
   paste it into another image (the destination) attempting to match the
   two images so that the pasting is non-obvious. This is
   known as a "seamless clone".

   The basic ideas are as follows:

   1) Figure out the interior and border of the source image
   2) Use the values of the border pixels in the destination image 
      as boundary conditions for solving a Poisson equation that tells
      us how to blend the images.
   
      No pixels from the destination except pixels on the border
      are used to compute the match.

   Solving the Poisson Equation
   ============================

   There are multiple ways to solve this equation - we choose an iterative
   method - specifically the Jacobi method. Iterative methods start with
   a guess of the solution and then iterate to try and improve the guess
   until it stops changing.  If the problem was well-suited for the method
   then it will stop and where it stops will be the solution.

   The Jacobi method is the simplest iterative method and converges slowly - 
   that is we need a lot of iterations to get to the answer, but it is the
   easiest method to write.

   Jacobi Iterations
   =================

   Our initial guess is going to be the source image itself.  This is a pretty
   good guess for what the blended image will look like and it means that
   we won't have to do as many iterations compared to if we had started far
   from the final solution.

   ImageGuess_prev (Floating point)
   ImageGuess_next (Floating point)

   DestinationImg
   SourceImg

   Follow these steps to implement one iteration:

   1) For every pixel p in the interior, compute two sums over the four neighboring pixels:
      Sum1: If the neighbor is in the interior then += ImageGuess_prev[neighbor]
             else if the neighbor in on the border then += DestinationImg[neighbor]

      Sum2: += SourceImg[p] - SourceImg[neighbor]   (for all four neighbors)

   2) Calculate the new pixel value:
      float newVal= (Sum1 + Sum2) / 4.f  <------ Notice that the result is FLOATING POINT
      ImageGuess_next[p] = min(255, max(0, newVal)); //clamp to [0, 255]


    In this assignment we will do 800 iterations.
   */
   
#include "utils.h"
#include <algorithm>

// get 2d position from block
__device__
int2 get2dPos() { 
	return make_int2(
		blockIdx.x * blockDim.x + threadIdx.x,
       	blockIdx.y * blockDim.y + threadIdx.y
	);
}

// check whether a a value is within the image bounds
__device__
bool withinBounds(const int x, const int y, const size_t numRowsSource, const size_t numColsSource) {
    return ((x < numColsSource) && (y < numRowsSource));
}

__device__
bool masked(uchar4 val) {
	return (val.x != 255 || val.y != 255 || val.z != 255);
}

__device__
int getm(int x, int y, size_t numColsSource) {
	return y*numColsSource + x;
}

__global__
void maskPredicateKernel(
	const uchar4* const d_sourceImg,
	int* d_borderPredicate,
	int* d_interiorPredicate,
	const size_t numRowsSource,
	const size_t numColsSource) {
	
    const int2 p = get2dPos();
	const int  m = getm(p.x, p.y, numColsSource);
    
    if(!withinBounds(p.x, p.y, numRowsSource, numColsSource))
         return;

 	// run through each pixel and determine if its 
	// on the border, or if its on the interior border
	
	if(masked(d_sourceImg[m])) {
		int inbounds = 0;
		int interior = 0;

		// count how many of our neighbors are masked,
		// and how many neighbors we have
		if (withinBounds(p.x, p.y+1, numRowsSource, numColsSource)) {
			inbounds++;
			if(masked(d_sourceImg[getm(p.x, p.y+1, numColsSource)]))
				interior++;		
	
		}
		if (withinBounds(p.x, p.y-1, numRowsSource, numColsSource)) {
			inbounds++;
			if(masked(d_sourceImg[getm(p.x, p.y-1, numColsSource)]))
				interior++;		
	
		}
		if (withinBounds(p.x+1, p.y, numRowsSource, numColsSource)) {
			inbounds++;
			if(masked(d_sourceImg[getm(p.x+1, p.y, numColsSource)]))
				interior++;		
		}
		if (withinBounds(p.x-1, p.y, numRowsSource, numColsSource)) {
			inbounds++;
			if(masked(d_sourceImg[getm(p.x-1, p.y, numColsSource)]))
				interior++;		
		}

		// clear out the values so we don't
		// have to memset this destination stuff
		d_interiorPredicate[m] = 0;
		d_borderPredicate[m]   = 0;
	
		// if all our neighbors are masked, then its interior
		if(inbounds == interior) {
			d_interiorPredicate[m] = 1;
		} else if (interior > 0) {
			d_borderPredicate[m] = 1;
		}
	}
}

__global__
void separateChannelsKernel(
	const uchar4* const inputImageRGBA,
	float* const redChannel,
	float* const greenChannel,
	float* const blueChannel,
	size_t numRows,
	size_t numCols)
{
    const int2 p = get2dPos();
	const int  m = getm(p.x, p.y, numCols);
    
    if(!withinBounds(p.x, p.y, numRows, numCols))
         return;

  	redChannel[m]   = (float)inputImageRGBA[m].x;
  	greenChannel[m] = (float)inputImageRGBA[m].y;
  	blueChannel[m]  = (float)inputImageRGBA[m].z;
}

__global__
void recombineChannelsKernel(
	uchar4* outputImageRGBA,
	float* const redChannel,
	float* const greenChannel,
	float* const blueChannel,
	size_t numRows,
	size_t numCols)
{
    const int2 p = get2dPos();
	const int  m = getm(p.x, p.y, numCols);
    
    if(!withinBounds(p.x, p.y, numRows, numCols))
         return;
	
	outputImageRGBA[m].x = (char)redChannel[m];
	outputImageRGBA[m].y = (char)greenChannel[m];
	outputImageRGBA[m].z = (char)blueChannel[m];
}

__global__
void jacobiKernel(
	float* d_in,
	float* d_out,
	const int* d_borderPredicate,
	const int* d_interiorPredicate,
	float* d_source,
	float* d_dest,
	size_t numRows,
	size_t numCols)
{
    const int2 p = get2dPos();
	const int  m = getm(p.x, p.y, numCols);
    
    if(!withinBounds(p.x, p.y, numRows, numCols))
         return;

	// calculate these values as indicated in the videos

	int lm;
	if(d_interiorPredicate[m]==1) {
		float a = 0.f, b=0.f, c=0.0f, d=0.f;
		float sourceVal = d_source[m];

		if(withinBounds(p.x, p.y+1, numRows, numCols)) {
			d++;
			lm = getm(p.x, p.y+1, numCols);
			if(d_interiorPredicate[lm]==1) {
				a += d_in[lm];
			} else if(d_borderPredicate[lm]==1) {
				b += d_dest[lm];
			}
			c += (sourceVal-d_source[lm]);
		}
		
		if(withinBounds(p.x, p.y-1, numRows, numCols)) {
			d++;
			lm = getm(p.x, p.y-1, numCols);
			if(d_interiorPredicate[lm]==1) {
				a += d_in[lm];
			} else if(d_borderPredicate[lm]==1) {
				b += d_dest[lm];
			}
			c += (sourceVal-d_source[lm]);
		}
		
		if(withinBounds(p.x+1, p.y, numRows, numCols)) {
			d++;
			lm = getm(p.x+1, p.y, numCols);
			if(d_interiorPredicate[lm]==1) {
				a += d_in[lm];
			} else if(d_borderPredicate[lm]==1) {
				b += d_dest[lm];
			}
			c += (sourceVal-d_source[lm]);
		}
		
		if(withinBounds(p.x-1, p.y, numRows, numCols)) {
			d++;
			lm = getm(p.x-1, p.y, numCols);
			if(d_interiorPredicate[lm]==1) {
				a += d_in[lm];
			} else if(d_borderPredicate[lm]==1) {
				b += d_dest[lm];
			}
			c += (sourceVal-d_source[lm]);
		}
		
		d_out[m] = min(255.f, max(0.0, (a + b + c)/d));
	} else {
		d_out[m] = d_dest[m];
	}
	
}

void your_blend(const uchar4* const h_sourceImg,  //IN
                const size_t numRowsSource, const size_t numColsSource,
                const uchar4* const h_destImg, //IN
                uchar4* const h_blendedImg) //OUT
{
	// first push the dest and source onto the gpu
	size_t imageSize = numRowsSource*numColsSource*sizeof(uchar4);
	
	uchar4* d_sourceImg;
	uchar4* d_destImg;
	uchar4* d_finalImg;

	checkCudaErrors(hipMalloc(&d_sourceImg, imageSize));
	checkCudaErrors(hipMalloc(&d_destImg, 	 imageSize));
	checkCudaErrors(hipMalloc(&d_finalImg,  imageSize));

  	checkCudaErrors(hipMemcpy(d_sourceImg, h_sourceImg, imageSize, hipMemcpyHostToDevice));
  	checkCudaErrors(hipMemcpy(d_destImg, 	h_destImg, 	 imageSize, hipMemcpyHostToDevice));

	// allocate predicate stuff
	size_t predicateSize = numRowsSource*numColsSource*sizeof(int);
	int* d_borderPredicate;
	int* d_interiorPredicate;

	checkCudaErrors(hipMalloc(&d_borderPredicate, 	 predicateSize));
	checkCudaErrors(hipMalloc(&d_interiorPredicate, predicateSize));

	// make reusable dims
	const dim3 blockSize(32, 32);
    const dim3 gridSize(numColsSource/blockSize.x + 1, numRowsSource/blockSize.y + 1);


	/**
     1) Compute a mask of the pixels from the source image to be copied
        The pixels that shouldn't be copied are completely white, they
        have R=255, G=255, B=255.  Any other pixels SHOULD be copied.
	**/

	/**
     2) Compute the interior and border regions of the mask.  An interior
        pixel has all 4 neighbors also inside the mask.  A border pixel is
        in the mask itself, but has at least one neighbor that isn't.
	**/

	// generate the predicates
	maskPredicateKernel<<<gridSize, blockSize>>>(
		d_sourceImg,
		d_borderPredicate,
		d_interiorPredicate,
		numRowsSource,
		numColsSource
	);

 	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

 	/**
     3) Separate out the incoming image into three separate channels
	**/
	size_t floatSize = numRowsSource*numColsSource*sizeof(float);
	float *d_sourceImgR, *d_sourceImgG, *d_sourceImgB; 
	float *d_destImgR,   *d_destImgG, 	*d_destImgB;

	checkCudaErrors(hipMalloc(&d_sourceImgR, floatSize));
	checkCudaErrors(hipMalloc(&d_sourceImgG, floatSize));
	checkCudaErrors(hipMalloc(&d_sourceImgB, floatSize));
	
	checkCudaErrors(hipMalloc(&d_destImgR, floatSize));
	checkCudaErrors(hipMalloc(&d_destImgG, floatSize));
	checkCudaErrors(hipMalloc(&d_destImgB, floatSize));
	
	separateChannelsKernel<<<gridSize, blockSize>>>(
		d_sourceImg,
		d_sourceImgR,
		d_sourceImgG,
		d_sourceImgB,
		numRowsSource,
		numColsSource);

 	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

	separateChannelsKernel<<<gridSize, blockSize>>>(
		d_destImg,
		d_destImgR,
		d_destImgG,
		d_destImgB,
		numRowsSource,
		numColsSource);

 	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

	/** 
     4) Create two float(!) buffers for each color channel that will
        act as our guesses.  Initialize them to the respective color
        channel of the source image since that will act as our intial guess.
	**/
	
	// allocate floats
	float *d_r0, *d_r1, *d_g0, *d_g1, *d_b0, *d_b1; 
	checkCudaErrors(hipMalloc(&d_r0, floatSize));
	checkCudaErrors(hipMalloc(&d_r1, floatSize));
	checkCudaErrors(hipMalloc(&d_b0, floatSize));
	checkCudaErrors(hipMalloc(&d_b1, floatSize));
	checkCudaErrors(hipMalloc(&d_g0, floatSize));
	checkCudaErrors(hipMalloc(&d_g1, floatSize));


  	checkCudaErrors(hipMemcpy(d_r0, d_sourceImgR, floatSize, hipMemcpyDeviceToDevice));
  	checkCudaErrors(hipMemcpy(d_g0, d_sourceImgG, floatSize, hipMemcpyDeviceToDevice));
  	checkCudaErrors(hipMemcpy(d_b0, d_sourceImgB, floatSize, hipMemcpyDeviceToDevice));

 	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

	/**
     5) For each color channel perform the Jacobi iteration described 
        above 800 times.
	**/
	for(int i = 0; i < 800; i++) {
		jacobiKernel<<<gridSize, blockSize>>>(
			d_r0, 
			d_r1,
			d_borderPredicate,
			d_interiorPredicate,
			d_sourceImgR,
			d_destImgR,
			numRowsSource,
			numColsSource
		);
		std::swap(d_r0, d_r1);

		jacobiKernel<<<gridSize, blockSize>>>(
			d_g0, 
			d_g1,
			d_borderPredicate,
			d_interiorPredicate,
			d_sourceImgG,
			d_destImgG,
			numRowsSource,
			numColsSource
		);
		std::swap(d_g0, d_g1);

		jacobiKernel<<<gridSize, blockSize>>>(
			d_b0, 
			d_b1,
			d_borderPredicate,
			d_interiorPredicate,
			d_sourceImgB,
			d_destImgB,
			numRowsSource,
			numColsSource
		);
		std::swap(d_b0, d_b1);
	}

	/**
     6) Create the output image by replacing all the interior pixels
        in the destination image with the result of the Jacobi iterations.
        Just cast the floating point values to unsigned chars since we have
        already made sure to clamp them to the correct range.
	**/

	// lets assume that d_r0, d_g0, d_b0 are the final pass
	recombineChannelsKernel<<<gridSize, blockSize>>>(
		d_finalImg,
		d_r0,
		d_g0,
		d_b0,
		numRowsSource,
		numColsSource);

 	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

	// copy device final image to host
  	checkCudaErrors(hipMemcpy(h_blendedImg, d_finalImg, imageSize, hipMemcpyDeviceToHost));

	// cleanup
  	checkCudaErrors(hipFree(d_sourceImg));
  	checkCudaErrors(hipFree(d_destImg));
	checkCudaErrors(hipFree(d_finalImg));

	checkCudaErrors(hipFree(d_sourceImgR));
	checkCudaErrors(hipFree(d_sourceImgG));
	checkCudaErrors(hipFree(d_sourceImgB));

	checkCudaErrors(hipFree(d_destImgR));
	checkCudaErrors(hipFree(d_destImgG));
	checkCudaErrors(hipFree(d_destImgB));

	checkCudaErrors(hipFree(d_r0));
	checkCudaErrors(hipFree(d_r1));
	checkCudaErrors(hipFree(d_g0));
	checkCudaErrors(hipFree(d_g1));
	checkCudaErrors(hipFree(d_b0));
	checkCudaErrors(hipFree(d_b1));
}
